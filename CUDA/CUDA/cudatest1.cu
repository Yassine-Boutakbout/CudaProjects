#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"

__global__ void test() {
	//Empty Kernel
	printf("hello from CUDA");
}

int main() {
	test << <1, 1 >> > ();
	return 0;
}