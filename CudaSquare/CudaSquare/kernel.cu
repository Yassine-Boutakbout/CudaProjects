#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include "malloc.h"


__global__ void kernel(int* A, int N) {

	int idx = blockDim.x * blockIdx.x + threadIdx.x;
	if (idx < N)
		A[idx] = A[idx] * A[idx];
}

int main() {

	int N = 3;
	size_t size = N * sizeof(int);

	//allocation input vector
	int* H_A = (int*)malloc(N * sizeof(int));


	// Initialize input vectors
	for (int i = 0; i < N; i++)
	{
		H_A[i] = i;
	}

	int* D_A;

	// Allocate vectors in device memory
	hipMalloc(&D_A, size);

	// Copy vector from host memory to device memory
	hipMemcpy(D_A, H_A, size, hipMemcpyHostToDevice);

	int threads = 8;
	int blocks = (N + 7) / 8;

	//set the grid & block sizes
	dim3 gridDim(blocks);//gridDim nbr of blocks in a grid
	dim3 blockDim(threads);//blockDim nbr threads in a block

	//Invoke kernel
	kernel << <gridDim, blockDim >> > (D_A,N);

	// Copy result from device memory to host memory
	hipMemcpy(H_A, D_A, size, hipMemcpyDeviceToHost);

	for (int j = 0; j < N; j++)
	{
		printf("A[%d]=%d \n", j, H_A[j]);
	}

	hipFree(D_A);
	free(H_A);
}