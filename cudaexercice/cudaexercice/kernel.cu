#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include"stdio.h"

//prototypes

__global__ void mycourse(char*);

//host function
int main(int argc, char** argv) {
	//desired output
	char str[] = "my cuda course!";
	//allocate memory on the device
	char* d_str;
	size_t size = sizeof(str);
	hipMalloc((void**)&d_str, size);

	//copy the string of the device
	hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);

	//set the grid & block sizes
	dim3 gridDim(3);//gridDim nbr of blocks in a grid -here one block per word, block 1D
	dim3 blockDim(6);//blockDim nbr threads in a block - here one thread per character


	hipSetDevice(0);
	//invoke the kernel
	mycourse <<<gridDim, blockDim >>> (d_str);//execution of the GPU fct

	hipDeviceSynchronize();
	//free up the allocated memory on the device
	hipFree(d_str);
	hipDeviceReset();
	return 0;
}

//device kernel
__global__ void mycourse(char* str) {

	//determine where in the thread grid we are  0 1 2*5 + 0 1 2 3 4
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//print content of string
	printf("%c", str[idx]);
}