#include "hip/hip_runtime.h"

#include "hip/device_functions.h"
#include"stdio.h"

__global__ void matrixMult(int* a, int* b, int* c, int width) {
	int i, sum = 0;
	int col = threadIdx.x + blockDim.x * blockIdx.x;
	int row = threadIdx.y + blockDim.y * blockIdx.y;
	if (col < width && row < width) {
		for (i = 0; i < width; i++) {
			sum += a[row * width + i] * b[i * width + col];
		}
		c[row * width + col] = sum;
	}
}

int main(int argc, char* argv[]) {
	int N = 3;
	//int a[N][N], b[N][N], c[N][N];
	int* h_a, * h_b, * h_c;
	int* dev_a, * dev_b, * dev_c;
	int i, j;
	int size = N * N * sizeof(int);
	hipHostMalloc((void**)&h_a, size, hipHostMallocDefault);
	hipHostMalloc((void**)&h_b, size, hipHostMallocDefault);
	hipHostMalloc((void**)&h_c, size, hipHostMallocDefault);

	// initialize matrices a and b with appropriate values
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			h_a[i * N + j] = i + j;
			h_b[i * N + j] = i * j;

		}
	}
	printf(" ****matrice 1***** \n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			printf("%d\t", h_a[i * N + j]);

		}
		printf("\n");
	}
	printf(" ****matrice 2*****\n");
	for (i = 0; i < N; i++) {
		for (j = 0; j < N; j++) {
			printf("%d\t", h_b[i * N + j]);
		}
		printf("\n");
	}

	hipMalloc((void**)&dev_a, size);
	hipMalloc((void**)&dev_b, size);
	hipMalloc((void**)&dev_c, size);

	hipMemcpy(dev_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(dev_b, h_b, size, hipMemcpyHostToDevice);

	dim3 gridDim(1, 1);
	dim3 blockDim(N, N);

	matrixMult <<<gridDim, blockDim >>> (dev_a, dev_b, dev_c, N);

	hipMemcpy(h_c, dev_c, size, hipMemcpyDeviceToHost);
	printf("****R�sultat****\n");
	for (i = 0; i < N; i++)
	{
		for (int j = 0; j < N; j++)
		{
			printf("%d\t", h_c[i * N + j]);
		}
		printf("\n");
	}

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);

	return 0;
}