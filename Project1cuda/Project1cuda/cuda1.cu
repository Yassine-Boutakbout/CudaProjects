#include"hip/hip_runtime.h"

#include"stdio.h"

//prototypes

__global__ void helloWorld(char*);

//host function

int main(int argc, char** argv) {
	int i;
	//desired output
	char str[] = "Hello World!";
	//mangle content of output
	for (i = 0; i < 12; i++)
	{
		//allocate memory on the device
		char* d_str;
		size_t size = sizeof(str);
		hipMalloc((void**)&d_str, size);
		//copy the string of the device
		hipMemcpy(d_str, str, size, hipMemcpyHostToDevice);
		//set the grid & block sizes
		dim3 gridDim(2);//gridDim nbr of blocks in a grid -here one block per word, block 1D
		dim3 blockDim(6);//blockDim nbr threads in a block - here one thread per character
		//invoke the kernel
		helloWorld<<<gridDim, blockDim >>> (d_str);//execution of the GPU fct
		//retrieve the results from the device
		hipMemcpy(str, d_str, size, hipMemcpyDeviceToHost);
		//free up the allocated memory on the device
		hipFree(d_str);
		//result
		printf("%s \n", str);
		return 0;
	}
}

//device kernel

__global__ void helloWorld(char* str) {

	//determine where in the thread grid wz are
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	//unmangle output
	str[idx] += idx;
}