#include <stdlib.h>
#include "hip/hip_runtime.h"

#include "stdio.h"
#include "malloc.h"


const int N = 2;
const int blocksize = 1;
const int MAX = 100;

__host__ void add_matrix_cpu(float* a, float* b, float* c, int N) {
	int i, j;
	printf("Host \n");
	for (i = 0; i < N; i++)
	{
		printf("\n");
		for (j = 0; j < N; j++)
		{
			c[i * N + j] = a[i * N + j] + b[i * N + j];
			printf("C[%d]=%f \t",i*N+1,c[i * N + 1]);
		}
	}
}

__global__ void add_matrix(float* a, float* b, float* c, int N) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int index = i + j * N;
	if (i < N && j < N)
		c[index] = a[index] + b[index];
}

int main() {
	int k;
	float* a = new float[N * N];
	float* b = new float[N * N];
	float* c = new float[N * N];


	for (int i = 0; i < N * N; ++i) {
		a[i] = 1.0f; b[i] = 3.5f;
	}
	float* ad, * bd, * cd;
	const int size = N * N * sizeof(float);
	hipMalloc((void**)&ad, size);
	hipMalloc((void**)&bd, size);
	hipMalloc((void**)&cd, size);
	hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
	hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
	dim3 dimBlock(blocksize, blocksize);
	dim3 dimGrid(N / dimBlock.x, N / dimBlock.y);

	add_matrix <<<dimGrid, dimBlock >>> (ad, bd, cd, N);
	hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
	hipFree(ad);
	hipFree(bd);
	hipFree(cd);
	printf("Device \n");
	for (int m = 0; m < 4; m++)
	{
		if (m % 2 == 0)
		{
			printf("\n");
		}
		printf("c[%d]=%f \t",m,c[m]);
	}
	

	for (k = 1; k <= MAX; k++)
		add_matrix_cpu(a, b, c, N);
	delete[] a;
	delete[] b;
	delete[] c;
}