#include"hip/hip_runtime.h"

#include"stdio.h"
#include <malloc.h>
#define N (1024*1024)
#define M (5000)

__global__ void cudakernel(float* buf)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	buf[i] = 1.0f * i / N;
	for (int j = 0; j < M; j++) { buf[i] = buf[i] * buf[i] - 0.25f; }
}

int main()
{
	float *data;
	float *d_data;
	data = (float*)malloc(N*sizeof(float));
	hipMalloc(&d_data, N * sizeof(float));
	hipMemcpy(d_data, data, N * sizeof(float), hipMemcpyHostToDevice);
	cudakernel<<< N/256, 256 >>>(d_data);
	hipMemcpy(data, d_data, N * sizeof(float), hipMemcpyDeviceToHost);
	hipFree(d_data);

	int sel;
	sel = 1000;
	printf("data[%d]=%f \n", sel, data[sel]);
}