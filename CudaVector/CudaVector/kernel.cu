#include "hip/hip_runtime.h"

#include "stdio.h"
#include "malloc.h"

__global__ void vector(int* X, int* Y,int* Z,int sc, int N) {

	int idx= blockDim.x * blockIdx.x + threadIdx.x;

	if (idx < N)
	{
		Z[idx] = sc * X[idx] + Y[idx];
		printf("Z[%d]=%d \n", idx, Z[idx]);
	}
}

int main() {

	int N = 16, sc = 2;
	size_t size = N * sizeof(int);

	//Allocate input vectors h_X and h_Y in host memory
	int* H_X = (int*)malloc(size);
	int* H_Y = (int*)malloc(size);
	int* H_Z = (int*)malloc(size);

	// Initialize input vectors
	for (int i = 0; i < N; i++)
	{
		H_X[i] = 1;
		H_Y[i] = 2;
	}

	int *D_X,*D_Y,*D_Z;

	// Allocate vectors in device memory
	hipMalloc(&D_X, size);
	hipMalloc(&D_Y, size);
	hipMalloc(&D_Z, size);


	// Copy vectors from host memory to device memory
	hipMemcpy(D_X, H_X, size, hipMemcpyHostToDevice);
	hipMemcpy(D_Y, H_Y, size, hipMemcpyHostToDevice);

	int threads = 8;
	int blocks = (N+7)/ 8;

	//set the grid & block sizes
	dim3 gridDim(blocks);//gridDim nbr of blocks in a grid
	dim3 blockDim(threads);//blockDim nbr threads in a block

	//Invoke kernel
	vector <<<gridDim, blockDim >>> (D_X, D_Y, D_Z, sc, N);

	// Copy result from device memory to host memory
	hipMemcpy(H_Z, D_Z, size, hipMemcpyDeviceToHost);

	// Free device memory
	hipFree(D_X);
	hipFree(D_Y);
	hipFree(D_Z);

	// Free host memory
	free(H_X);
	free(H_Y);
	free(H_Z);
}